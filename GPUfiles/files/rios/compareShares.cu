#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <iostream>
#include <fstream>
#include <chrono>
#include <math.h>

typedef std::chrono::high_resolution_clock Time;
using namespace std::chrono;

int main(){

	std::ifstream infile("stocks2.csv");
	int aV,mV;
	thrust::host_vector<double> cpu_values;
	while (infile >> aV >> mV){
		int dif = aV-mV;
		cpu_values.push_back(sqrt((dif*dif)));
	}
	Time::time_point t1 = Time::now();
	thrust::device_vector<double> gpu_values (cpu_values);
	Time::time_point t2 = Time::now();
	duration<double> time = duration_cast<duration<double>>(t2 - t1);
	std::cerr <<"Time: "<<"\n"<< time.count() << '\n';

	int sharesDifMean = thrust::reduce(gpu_values.begin(), gpu_values.end(), (int) 1, thrust::multiplies<int>());
	std::cout << "MEDIA: " << sharesDifMean <<"\n"; 
